#include <cudatbx/cuda_base.cuh>
#include <simtbx/gpu/structure_factors.h>

namespace simtbx {
namespace gpu {

  gpu_energy_channels::gpu_energy_channels(int const& deviceId){
    h_deviceID = deviceId;
    hipSetDevice(deviceId);
  }

  void 
  gpu_energy_channels::structure_factors_to_GPU_detail(af::shared<double> linear_amplitudes){
    double * raw_ptr = linear_amplitudes.begin();
    CUDAREAL * cu_Fhkl = NULL;
    cudaSafeCall(hipMalloc((void ** )&cu_Fhkl, 
                     sizeof(*cu_Fhkl) * linear_amplitudes.size()));
    cudaSafeCall(hipMemcpy(cu_Fhkl, raw_ptr, 
                     sizeof(*cu_Fhkl) * linear_amplitudes.size(), hipMemcpyHostToDevice));

    d_channel_Fhkl.push_back(cu_Fhkl);
  }

  void gpu_energy_channels::free_detail(){
        cudaSafeCall(hipSetDevice(h_deviceID));
        for (int i_cu_ptr=0; i_cu_ptr < d_channel_Fhkl.size(); ++i_cu_ptr){
          cudaSafeCall(hipFree(d_channel_Fhkl[i_cu_ptr]));
        }
  }
} // gpu
} // simtbx
